
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

void check_cuda(hipError_t error, const char *filename, const int line)
{
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line,
                 hipGetErrorName(error), hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__)

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  int x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }

  fclose(fp);
  return img;
}

__global__ void histogram_kernel() {
  printf("Warning: histogram_kernel not implemented!\n");
}

double Histogram(PPMImage *image, float *h_h) {
  float ms;
  hipEvent_t start, stop;

  // Create Events
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  // Launch kernel and compute kernel runtime.
  // Warning: make sure only the kernel is being profiled, memcpies should be
  // out of this region.
  CUDACHECK(hipEventRecord(start));
  histogram_kernel<<<1, 1>>>();
  CUDACHECK(hipEventRecord(stop));
  CUDACHECK(hipEventSynchronize(stop));
  CUDACHECK(hipEventElapsedTime(&ms, start, stop));

  // Destroy events
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));

  return ((double)ms) / 1000.0;
}

int main(int argc, char *argv[]) {

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return 1;
  }

  PPMImage *image = readPPM(argv[1]);
  float *h = (float *)malloc(sizeof(float) * 64);

  // Initialize histogram
  for (int i = 0; i < 64; i++)
    h[i] = 0.0;

  // Compute histogram
  double t = Histogram(image, h);

  for (int i = 0; i < 64; i++)
    printf("%0.3f ", h[i]);
  printf("\n");

  fprintf(stderr, "%lf\n", t);
  free(h);
}
