#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <chrono>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <optional>
#include <stdexcept>
#include <string>
#include <type_traits>

/** Marker for conditional expressions that are unlikely to happen. */
#define unlikely(condition) (__builtin_expect(!!(condition), false))
/** C-like restrict keyword. */
#define restrict __restrict__

template <typename Num, class = std::enable_if_t<std::is_integral_v<Num>>>
/** Does 'a * b' if possible, or returns 'nullopt' if an overflow or underflow would occur. */
static constexpr inline std::optional<Num> checked_mul(const Num a, const Num b) noexcept {
  if (b == 0) {
    return a * b;
  }

  constexpr Num MINUMUM = std::numeric_limits<Num>::min();
  constexpr Num MAXIMUM = std::numeric_limits<Num>::max();
  const Num lo = std::min(MINUMUM / b, MAXIMUM / b);
  const Num hi = std::max(MINUMUM / b, MAXIMUM / b);

  if unlikely (a < lo || a > hi) {
    return std::nullopt;
  }
  return a * b;
}

/** CUDA helper functions. */
namespace cuda {
  /** Number of threads per block. */
  static constexpr unsigned BLOCK_SIZE = THREADS_PER_BLOCK;
  static_assert(BLOCK_SIZE > 0);
  static_assert(BLOCK_SIZE % 32 == 0);

  /** Reports a hipError_t. */
  class error final : public std::runtime_error {
  private:
    explicit error(hipError_t errnum) : std::runtime_error(hipGetErrorString(errnum)) {}

    /** Always fails with 'cuda::error(errnum)'.
     *
     * Reduces code bloat from inlined 'cuda::error::check(result)' calls.
     */
    [[gnu::cold, noreturn]] static void fail(const hipError_t errnum) {
      throw error(errnum);
    }

  public:
    /** Throws an error if 'result' is not 'hipSuccess'.  */
    [[gnu::hot]] static inline void check(const hipError_t result) {
      if unlikely (result != hipSuccess) {
        fail(result);
      }
    }
  };

  namespace last_error {
    /** Removes that last error value and set it to 'hipSuccess'. */
    static void clear() noexcept {
      hipGetLastError();
    }

    /** Throws an error if 'hipGetLastError' returns something other than 'hipSuccess'. */
    static void check() {
      error::check(hipGetLastError());
    }
  }; // namespace last_error

  /** Checked 'hipDeviceSynchronize'. */
  static void device_synchronize() {
    error::check(hipDeviceSynchronize());
  }

  /** Calculate the number of blocks for 'count' elements. */
  static constexpr inline unsigned blocks(const unsigned count) noexcept {
    if unlikely (count == 0) {
      return 0;
    } else {
      // ceil(count / BLOCK_SIZE)
      return (count - 1) / BLOCK_SIZE + 1;
    }
  }

  /** Rounds 'count' to the nearest multiple of 'BLOCK_SIZE'. */
  static constexpr inline unsigned nearest_block_multiple(const unsigned count) {
    const auto multiple = checked_mul(BLOCK_SIZE, blocks(count));
    if unlikely (!multiple.has_value()) {
      throw std::length_error("array is too big");
    }
    return *multiple;
  }

  template <typename T>
  /** Size in bytes for an array of 'count' elements of 'T'. */
  static constexpr inline size_t byte_size(const unsigned count) noexcept {
    constexpr size_t max_count = std::numeric_limits<unsigned>::max();
    // this guarantees that 'count * sizeof(T)' cannot overflow
    static_assert(checked_mul<size_t>(max_count, sizeof(T)).has_value());

    return static_cast<size_t>(count) * sizeof(T);
  }

  /** Memory context, associated with a Function Execution Space Specifier. */
  enum context : bool {
    /** __host__ space (usually the CPU and main memory context). */
    host,
    /** __device__ space (one or more GPUs contexts). */
    device
  };

  template <typename T, context ctx>
  /** Allocates memory in 'ctx' for exactly 'count' elements of 'T'. */
  static T *malloc_exact(const unsigned count) {
    T *ptr = nullptr;

    if (ctx == context::device) {
      error::check(hipMalloc(&ptr, byte_size<T>(count)));
    } else {
      error::check(hipHostMalloc(&ptr, byte_size<T>(count)));
    }
    return ptr;
  }

  template <typename T, context ctx>
  /** Zeroes memory for 'count' elements of 'T' in 'ptr'. */
  static void memset_zero(T *ptr, const unsigned count) noexcept(ctx == context::host) {
    if (ctx == context::device) {
      error::check(hipMemset(ptr, 0, byte_size<T>(count)));
    } else {
      memset(ptr, 0, byte_size<T>(count));
    }
  }

  template <typename T, context ctx>
  /** Allocate 'count' elements of 'T' in the 'ctx'. */
  [[gnu::malloc]] static T *malloc(const unsigned count) {
    // the number of elements is rounded so that the size is evenly divisible by BLOCK_SIZE
    const unsigned closest_count = nearest_block_multiple(count);

    T *ptr = malloc_exact<T, ctx>(closest_count);
    // set the unused elements to zero
    if (count < closest_count) {
      memset_zero<T, ctx>(&ptr[count], closest_count - count);
    }
    return ptr;
  }

  template <typename T, context ctx>
  /** Allocate 'count' elements of 'T' in the 'ctx'. */
  [[gnu::malloc]] static T *calloc(const unsigned count) {
    // the number of elements is rounded so that the size is evenly divisible by BLOCK_SIZE
    const unsigned closest_count = nearest_block_multiple(count);

    T *ptr = malloc_exact<T, ctx>(closest_count);
    memset_zero<T, ctx>(ptr, closest_count);
    return ptr;
  }

  template <typename T, context ctx>
  /** Release memory allocated in 'cuda::malloc'. */
  static void free(T *ptr) {
    if (ctx == context::device) {
      error::check(hipFree(ptr));
    } else {
      error::check(hipHostFree(ptr));
    }
  }

  template <context src_ctx, context dst_ctx>
  /** Dictates the kind of memcpy used in 'hipMemcpy', given source and destination contexts. */
  static constexpr hipMemcpyKind memcpy_kind() noexcept {
    switch (src_ctx) {
    case context::host:
      switch (dst_ctx) {
      case context::host:
        return hipMemcpyHostToHost;
      case context::device:
        return hipMemcpyHostToDevice;
      default:
        return hipMemcpyDefault;
      }
    case context::device:
      switch (dst_ctx) {
      case context::host:
        return hipMemcpyDeviceToHost;
      case context::device:
        return hipMemcpyDeviceToDevice;
      default:
        return hipMemcpyDefault;
      }
    default:
      return hipMemcpyDefault;
    }
  }

  template <typename T, context dst_ctx, context src_ctx>
  /** Copies 'count' elements of 'T' from 'src' to 'dst', given their contexts. */
  static void memcpy(T *dst, const T *src, const unsigned count) {
    error::check(hipMemcpy(dst, src, byte_size<T>(count), memcpy_kind<src_ctx, dst_ctx>()));
  }

  template <typename T, context ctx = cuda::context::host>
  /** A CUDA Execution-Space aware smart pointer that behaves like an array of 'T'. */
  class array final {
  private:
    const unsigned size_;
    T *const data_;

    explicit array(const unsigned size, T *const data) : size_(size), data_(data) {}

  public:
    /** Allocates an array of 'size' elements. */
    explicit array(const unsigned count) : array(count, malloc<T, ctx>(count)) {}

    /** Prevent implicit copies. */
    array(array<T, ctx> &) = delete;
    array(const array<T, ctx> &) = delete;
    /** Moves should still be okay. */
    constexpr array(array<T, ctx> &&) noexcept = default;

    /** Copies data from another array, possibly in another context. */
    static array<T, ctx> zeroed(const unsigned count) {
      return array<T, ctx>(count, calloc<T, ctx>(count));
    }

    template <context other>
    /** Copies data from another array, possibly in another context. */
    static array<T, ctx> copy_from(const array<T, other> &source) {
      auto dst = array<T, ctx>(source.size());
      memcpy<T, ctx, other>(dst.data(), source.data(), dst.size());
      return dst;
    }

    ~array() {
      free<T, ctx>(data());
    }

    /** Pointer to the underlying array. */
    constexpr T *data() noexcept {
      return data_;
    }
    constexpr const T *data() const noexcept {
      return data_;
    }

    /** Array size. */
    constexpr unsigned size() const noexcept {
      return size_;
    }

    inline T *begin() noexcept {
      static_assert(ctx == context::host);
      return data();
    }
    inline const T *begin() const noexcept {
      static_assert(ctx == context::host);
      return data();
    }

    inline T *end() noexcept {
      static_assert(ctx == context::host);
      return data() + size();
    }
    inline const T *end() const noexcept {
      static_assert(ctx == context::host);
      return data() + size();
    }

    inline T &operator[](const unsigned index) noexcept {
      static_assert(ctx == context::host);
      return data()[index];
    }
    inline const T &operator[](const unsigned index) const noexcept {
      static_assert(ctx == context::host);
      return data()[index];
    }
  };

  /** Smart wrapper for 'hipEvent_t'. */
  class event final {
  private:
    hipEvent_t handle = nullptr;

    explicit event() {
      error::check(hipEventCreateWithFlags(&handle, hipEventDefault | hipEventBlockingSync));
    }

  public:
    /** Prevent implicit copies. */
    event(event &) = delete;
    event(const event &) = delete;
    /** Moves should still be okay. */
    constexpr event(event &&) noexcept = default;

    static event create() {
      return event();
    }

    ~event() {
      error::check(hipEventDestroy(handle));
      handle = nullptr;
    }

    void query() const {
      error::check(hipEventQuery(handle));
    }

    void record(hipStream_t stream = 0) {
      error::check(hipEventRecord(handle, stream));
    }

    void synchronize() {
      error::check(hipEventSynchronize(handle));
    }

    using milliseconds = std::chrono::duration<float, std::milli>;

    static milliseconds elapsed_time(const event &start, const event &end) {
      float ms = 0.0f;
      error::check(hipEventElapsedTime(&ms, start.handle, end.handle));
      return milliseconds(ms);
    }

    milliseconds elapsed_from(const event &start) const {
      return elapsed_time(start, *this);
    }

    milliseconds operator-(const event &start) const {
      return elapsed_from(start);
    }
  };
} // namespace cuda

/** Image utilities for PPM format. */
namespace PPM {
  /** A single pixel in a PPM image. */
  struct [[gnu::packed]] Pixel final {
  public:
    Pixel() = delete;

    /** Represents a single color in a pixel. */
    using Component = uint8_t;
    // each color component should be a single byte
    static_assert(sizeof(Component) == 1);

    Component red;
    Component green;
    Component blue;

    /** Number of color components in a pixel. */
    static constexpr unsigned components() noexcept {
      return (sizeof(Pixel::red) + sizeof(Pixel::green) + sizeof(Pixel::blue)) /
             sizeof(Pixel::Component);
    }

    /** Maximum value for a color component. */
    static constexpr unsigned component_color() noexcept {
      return std::numeric_limits<Component>::max();
    }
  };
  // each pixel must have its components tightly packed
  static_assert(sizeof(Pixel) == Pixel::components() * sizeof(Pixel::Component));

  /** Image implemented as an array of pixels. */
  struct Image final {
  private:
    /** Size for the allocated array, given the image dimensions. */
    static unsigned alloc_size(unsigned width, unsigned height) {
      const auto size = checked_mul(width, height);
      if unlikely (!size.has_value()) {
        throw std::bad_alloc();
      }
      return *size;
    }

    cuda::array<Pixel> content_;

    constexpr Pixel *data() noexcept {
      return content_.data();
    }

    /** Allocate a new image with 'width * height' pixels. */
    Image(const unsigned width, const unsigned height) : content_(alloc_size(width, height)) {}

  public:
    Image(Image &) = delete;
    Image(const Image &) = delete;
    constexpr Image(Image &&image) noexcept = default;

    /** The pixels that form the image. */
    constexpr const cuda::array<Pixel> &content() const noexcept {
      return content_;
    }

    /** Number of pixels in the image. */
    constexpr unsigned size() const noexcept {
      return content_.size();
    }

    /** Size in byte for all the image pixels. */
    constexpr std::streamsize bytes() const noexcept {
      return checked_mul<std::streamsize>(size(), sizeof(Pixel)).value();
    }

    /** Read a PPM image from file located at 'filename'. */
    static Image read(const char *filename) {
      auto file = std::ifstream();
      file.exceptions(std::ifstream::badbit | std::ifstream::failbit | std::ifstream::eofbit);
      file.open(filename, std::fstream::in);

      auto line = std::string();
      std::getline(file, line);
      if unlikely (line != "P6") {
        throw std::invalid_argument("Invalid image format (must be 'P6')");
      }

      constexpr auto max_size = std::numeric_limits<std::streamsize>::max();
      while (file.get() == '#') {
        file.ignore(max_size, '\n');
      }
      file.unget();

      unsigned width, height;
      unsigned component_color;
      file >> width >> height >> component_color;
      if unlikely (component_color != Pixel::component_color()) {
        throw std::invalid_argument("Image does not have 8-bits components");
      }
      file.ignore(max_size, '\n');

      auto image = Image(width, height);
      file.read(reinterpret_cast<char *>(image.data()), image.bytes());

      return image;
    }
  };
}; // namespace PPM

static __launch_bounds__(cuda::BLOCK_SIZE) __global__ void histogram_kernel() {
  printf("Warning: histogram_kernel not implemented!\n");
}

using seconds = std::chrono::duration<double>;

static seconds histogram(PPM::Image &image, std::array<double, histogram::SIZE> &h) {
  // Create Events
  auto start = cuda::event::create();
  auto stop = cuda::event::create();

  // Launch kernel and compute kernel runtime.
  // Warning: make sure only the kernel is being profiled, memcpies should be
  // out of this region.
  start.record();
  histogram_kernel<<<1, 1>>>();
  stop.record();
  stop.synchronize();

  return stop - start;
}

int main(const int argc, const char *const *const argv) {
  if unlikely (argc != 2) {
    throw std::invalid_argument("Error: missing path to input file\n");
    return EXIT_FAILURE;
  }

  const auto image = PPM::Image::read(argv[1]);
  auto h = cuda::array<float, cuda::host>(64);

  // Initialize histogram
  for (float &hi : h) {
    hi = 0.0;
  }

  // Compute histogram
  const auto elapsed = histogram(image, h);

  for (const float hi : h) {
    std::cout << std::fixed << std::setprecision(3) << hi << ' ';
  }
  std::cout << std::endl;

  std::cerr << std::fixed << elapsed.count() << std::endl;
  return EXIT_SUCCESS;
}
