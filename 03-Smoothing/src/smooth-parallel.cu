#include <hip/hip_runtime.h>
#include <omp.h>

#include <cstdio>
#include <iostream>
#include <stdexcept>

/** Marker for conditional expressions that are unlikely to happen. */
#define unlikely(condition) (__builtin_expect(!!(condition), false))
/** C-like restrict keyword. */
#define restrict __restrict__

static constexpr const char *COMMENT = "Histogram_GPU";
static constexpr unsigned RGB_COMPONENT_COLOR = 255;

static void check_cuda(hipError_t error, const char *filename, const int line) {
  if unlikely (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line, hipGetErrorName(error),
            hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__)

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  unsigned x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }

  fclose(fp);
  return img;
}

static void writePPM(PPMImage *img) {

  fprintf(stdout, "P6\n");
  fprintf(stdout, "# %s\n", COMMENT);
  fprintf(stdout, "%d %d\n", img->x, img->y);
  fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

  fwrite(img->data, 3 * img->x, img->y, stdout);
  fclose(stdout);
}

#define MASK_WIDTH 15

// Implement this!
static __launch_bounds__(1) __global__ void smoothing_kernel(void) {
  printf("Error: smoothing kernel not implemented!\n");
}

static void smoothing(PPMImage *restrict image, const PPMImage *restrict image_copy) {
  smoothing_kernel<<<1, 1>>>();
}

int main(const int argc, const char *const *const argv) {
  if unlikely (argc != 2) {
    throw std::invalid_argument("missing path to input file");
    return EXIT_FAILURE;
  }

  FILE *input = fopen(argv[1], "r");
  if unlikely (input == NULL) {
    fprintf(stderr, "Error: could not open input file!\n");
    return EXIT_FAILURE;
  }

  // Read input filename
  char filename[256];
  fscanf(input, "%255s\n", filename);

  // Read input file
  PPMImage *image = readPPM(filename);
  PPMImage *image_output = readPPM(filename);

  // Call Smoothing Kernel
  double t = omp_get_wtime();
  smoothing(image_output, image);
  t = omp_get_wtime() - t;

  // Write result to stdout
  writePPM(image_output);

  // Print time to stderr
  std::cerr << std::fixed << t << std::endl;

  // Cleanup
  free(image);
  free(image_output);

  return EXIT_SUCCESS;
}
