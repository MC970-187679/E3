
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define MASK_WIDTH 15

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

void check_cuda(hipError_t error, const char *filename, const int line)
{
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line,
                 hipGetErrorName(error), hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__)

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  int x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '6') {
    fprintf(stderr, "Invalid image format (must be 'P6')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  if (rgb_comp_color != RGB_COMPONENT_COLOR) {
    fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
    exit(1);
  }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
    fprintf(stderr, "Error loading image '%s'\n", filename);
    exit(1);
  }

  fclose(fp);
  return img;
}

void writePPM(PPMImage *img) {

  fprintf(stdout, "P6\n");
  fprintf(stdout, "# %s\n", COMMENT);
  fprintf(stdout, "%d %d\n", img->x, img->y);
  fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

  fwrite(img->data, 3 * img->x, img->y, stdout);
  fclose(stdout);
}

// Implement this!
__global__ void smoothing_kernel(void) {
  printf("Error: smoothing kernel not implemented!\n");
}

void Smoothing(PPMImage *image, PPMImage *image_copy) {
  smoothing_kernel<<<1, 1>>>();
}

int main(int argc, char *argv[]) {
  FILE *input;
  char filename[255];
  double t;

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return 1;
  }

  if ((input = fopen(argv[1], "r")) == NULL) {
    fprintf(stderr, "Error: could not open input file!\n");
    return 1;
  }

  // Read input filename
  fscanf(input, "%s\n", filename);

  // Read input file
  PPMImage *image = readPPM(filename);
  PPMImage *image_output = readPPM(filename);

  // Call Smoothing Kernel
  t = omp_get_wtime();
  Smoothing(image_output, image);
  t = omp_get_wtime() - t;

  // Write result to stdout
  writePPM(image_output);

  // Print time to stderr
  fprintf(stderr, "%lf\n", t);

  // Cleanup
  free(image);
  free(image_output);

  return 0;
}
