
#include <hip/hip_runtime.h>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <omp.h>

static __global__ void matrix_sum(/* ... */) {
  // TODO: Implement this kernel!
  printf("Hello, World from the GPU!\n");
}

int main(const int argc, const char *const *const argv) {
  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return EXIT_FAILURE;
  }

  FILE *input = fopen(argv[1], "r");
  if (input == NULL) {
    fprintf(stderr, "Error: could not open file\n");
    return EXIT_FAILURE;
  }

  // Input
  int rows = 0, cols = 0;
  assert(fscanf(input, "%d", &rows) == 1);
  assert(fscanf(input, "%d", &cols) == 1);
  fclose(input);

  // Allocate memory on the host
  int *A = new int[rows * cols];
  int *B = new int[rows * cols];
  int *C = new int[rows * cols];

  // Initialize memory
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      A[i * cols + j] = B[i * cols + j] = i + j;
    }
  }

  // Copy data to device
  // ...

  // Compute matrix sum on device
  // Leave only the kernel and synchronize inside the timing region!
  double t = omp_get_wtime();
  matrix_sum<<<1, 1>>>(/* ... */);
  hipDeviceSynchronize();
  t = omp_get_wtime() - t;

  // Copy data back to host
  // ...

  long long int sum = 0;

  // Keep this computation on the CPU
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      sum += C[i * cols + j];
    }
  }

  fprintf(stdout, "%lli\n", sum);
  fprintf(stderr, "%lf\n", t);

  delete[] A;
  delete[] B;
  delete[] C;

  return EXIT_SUCCESS;
}
